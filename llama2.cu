#include "hip/hip_runtime.h"
﻿/*
Inference for Llama-2 Transformer model in pure Cuda.
*/

#define _CRT_SECURE_NO_WARNINGS
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <math.h>
#include <string.h>
#include <time.h>
#include <hip/hip_runtime_api.h>
#include <hip/hip_fp16.h>
#include <hipcub/hipcub.hpp>

// ----------------------------------------------------------------------------
// GPU kernels

__global__ void element_wise_add_kernel(half* dest, half* src, int size) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < size)
        dest[i] = (half)((float)dest[i] + (float)src[i]);
}

__global__ void convert_fp32_to_fp16(half* out, float* in, int elements) {
    int index = blockIdx.x * 256 + threadIdx.x;
    if (index < elements)
        out[index] = (half)in[index];
}

__global__ void convert_fp16_to_fp32(float* out, half* in, int elements) {
    int index = blockIdx.x * 256 + threadIdx.x;
    if (index < elements)
        out[index] = (float)in[index];
}

// Single block - not enough parallelism for the GPU, but it's just 1% of total time
__global__ void rmsnorm_kernel(half* o, half* x, half* weight, int size, int elementsPerThread) {
    float ss = 0.0f;
    for (int i = 0; i < elementsPerThread; i++) {
        int index = threadIdx.x + i * 1024;
        if (index < size)
            ss += (float)x[index];
    }

    using BlockReduce = hipcub::BlockReduce<float, 1024>;
    __shared__ typename BlockReduce::TempStorage temp;
    ss = BlockReduce(temp).Sum(ss * ss);

    __shared__ float shared_ss;
    if (threadIdx.x == 0) {
        ss /= size;
        ss += 1e-5f;
        ss = 1.0f / sqrtf(ss);
        shared_ss = ss;
    }
    __syncthreads();
    ss = shared_ss;

    // normalize
    for (int i = 0; i < elementsPerThread; i++) {
        int index = threadIdx.x + i * 1024;
        if (index < size) {
            float val = (float)x[index];
            val *= ss * (float)weight[index];
            o[index] = (half)val;
        }
    }
}

// one output per warp so that we can parallelize the dot product across the warp
// Note that ~95% of total time is spent here, so optimizing this is important
__global__ void mat_vec_kernel(half* output, half* input, half* weight, int n, int d, int numSerialElements) {
    int index = blockIdx.x * blockDim.y + threadIdx.y;
    if (index >= d)
        return;

    float sum = 0;
    for (int i = 0; i < numSerialElements; i++) {
        int j = i * 32 + threadIdx.x;
        if (j < n)
            sum += ((float)weight[index * n + j]) * ((float)input[j]);
    }

    using WarpReduce = hipcub::WarpReduce<float>;
    __shared__ typename WarpReduce::TempStorage temp;
    sum = WarpReduce(temp).Sum(sum);

    if (threadIdx.x == 0)
        output[index] = (half)sum;
}

// Each block processes a single head
__global__ void RoPERotation_kernel(half* sq, half* sk, half* f_real, half* f_imag, int num_heads, int head_size) {
    int h = blockIdx.x;
    half* q = sq + h * head_size;
    half* k = sk + h * head_size;

    int i = threadIdx.x * 2;
    float q0 = q[i];
    float q1 = q[i + 1];
    float k0 = k[i];
    float k1 = k[i + 1];
    float fcr = f_real[i / 2];
    float fci = f_imag[i / 2];
    q[i] = q0 * fcr - q1 * fci;
    q[i + 1] = q0 * fci + q1 * fcr;
    k[i] = k0 * fcr - k1 * fci;
    k[i + 1] = k0 * fci + k1 * fcr;
}

__device__ void softmax_gpu(float* __restrict__ x, int size) {
    using BlockReduce = hipcub::BlockReduce<float, 1024>;
    __shared__ typename BlockReduce::TempStorage temp;
    __shared__ float shared_val;

    int tid = threadIdx.x;
    int step = blockDim.x;

    // find max value (for numerical stability)
    float max_val = tid < size ? x[tid] : 0;
    for (int i = tid + step; i < size; i += step)
        if (x[i] > max_val)
            max_val = x[i];

    max_val = BlockReduce(temp).Reduce(max_val, hipcub::Max());
    if (threadIdx.x == 0)
        shared_val = max_val;
    __syncthreads();
    max_val = shared_val;

    // exp and sum
    float sum = 0.0f;
    for (int i = tid; i < size; i += step) {
        x[i] = expf(x[i] - max_val);
        sum += x[i];
    }

    sum = BlockReduce(temp).Sum(sum);
    if (threadIdx.x == 0)
        shared_val = sum;
    __syncthreads();
    sum = shared_val;

    // normalize
    for (int i = tid; i < size; i += step)
        x[i] /= sum;
}

// Each block processes a single head
// Poor parallelism and even poorer memory access pattern.
// Ankan - TODO: optimize this.
#define MAX_SEQ_LEN 8192
__global__ void MultiHeadAttention_kernel(half* __restrict__ output, const half* __restrict__ sq,
    const half* __restrict__ key_cache, const half* __restrict__ value_cache,
    int num_heads, int head_size, int loff, int seq_len, int dim) {
    int h = blockIdx.x;

    // get the query vector for this head
    const half* q = sq + h * head_size;
    // attention scores for this head
    __shared__ float att[MAX_SEQ_LEN];

    // iterate over all timesteps, including the current one
    for (int t = threadIdx.x; t < seq_len; t += blockDim.x) {
        // get the key vector for this head and at this timestep
        const half* k = key_cache + loff + t * dim + h * head_size;
        // calculate the attention score as the dot product of q and k
        float score = 0.0f;
        for (int i = 0; i < head_size; i++)
            score += (float)q[i] * (float)k[i];
        score /= sqrtf(head_size);
        // save the score to the attention buffer
        att[t] = score;
    }
    __syncthreads();

    // softmax the scores to get attention weights
    softmax_gpu(att, seq_len);
    __syncthreads();

    // weighted sum of the values, store back into xb
    for (int i = threadIdx.x; i < head_size; i += blockDim.x) {
        float val = 0.0f;
        for (int t = 0; t < seq_len; t++)
            val += att[t] * (float)value_cache[loff + t * dim + h * head_size + i];
        output[h * head_size + i] = (half)val;
    }
}

__global__ void silu_element_wise_mul_kernel(half* dest, half* src, int size) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < size) {
        float val = (float)dest[i];
        val *= 1.0f / (1.0f + expf(-val));
        val *= (float)src[i];
        dest[i] = (half)val;
    }
}

// ----------------------------------------------------------------------------
// Transformer and RunState structs, and related memory management

typedef struct {
    int dim; // transformer dimension
    int hidden_dim; // for ffn layers
    int n_layers; // number of layers
    int n_heads; // number of query heads
    int n_kv_heads; // number of key/value heads (can be < query heads because of multiquery)
    int vocab_size; // vocabulary size, usually 256 (byte-level)
    int seq_len; // max sequence length
} Config;

typedef struct {
    // token embedding table
    half* token_embedding_table;    // (vocab_size, dim)
    // weights for rmsnorms
    half* rms_att_weight; // (layer, dim) rmsnorm weights
    half* rms_ffn_weight; // (layer, dim)
    // weights for matmuls
    half* wq; // (layer, dim, dim)
    half* wk; // (layer, dim, dim)
    half* wv; // (layer, dim, dim)
    half* wo; // (layer, dim, dim)
    // weights for ffn
    half* w1; // (layer, hidden_dim, dim)
    half* w2; // (layer, dim, hidden_dim)
    half* w3; // (layer, hidden_dim, dim)
    // final rmsnorm
    half* rms_final_weight; // (dim,)
    // freq_cis for RoPE relatively positional embeddings
    half* freq_cis_real; // (seq_len, dim/2)
    half* freq_cis_imag; // (seq_len, dim/2)
    // (optional) classifier weights for the logits, on the last layer
    half* wcls;
} TransformerWeights;

typedef struct {
    // current wave of activations
    half* x; // activation at current time stamp (dim,)
    half* xb; // same, but inside a residual branch (dim,)
    half* xb2; // an additional buffer just for convenience (dim,)
    half* hb; // buffer for hidden dimension in the ffn (hidden_dim,)
    half* hb2; // buffer for hidden dimension in the ffn (hidden_dim,)
    half* q; // query (dim,)
    half* k; // key (dim,)
    half* v; // value (dim,)
    half* logits_gpu; // output logits
    float* logits_temp; // logits in GPU memory converted to float
    float* logits; // logits copied CPU side
    // kv cache
    half* key_cache;   // (layer, seq_len, dim)
    half* value_cache; // (layer, seq_len, dim)
} RunState;

void malloc_run_state(RunState* s, Config* p) {
    hipMalloc((void**)&s->x, p->dim * sizeof(half));
    hipMalloc((void**)&s->xb, p->dim * sizeof(half));
    hipMalloc((void**)&s->xb2, p->dim * sizeof(half));
    hipMalloc((void**)&s->hb, p->hidden_dim * sizeof(half));
    hipMalloc((void**)&s->hb2, p->hidden_dim * sizeof(half));
    hipMalloc((void**)&s->q, p->dim * sizeof(half));
    hipMalloc((void**)&s->k, p->dim * sizeof(half));
    hipMalloc((void**)&s->v, p->dim * sizeof(half));
    hipMalloc((void**)&s->logits_gpu, p->vocab_size * sizeof(half));
    hipMalloc((void**)&s->key_cache, p->n_layers * p->seq_len * p->dim * sizeof(half));    // potentially huge allocs
    hipMalloc((void**)&s->value_cache, p->n_layers * p->seq_len * p->dim * sizeof(half));
    hipMalloc((void**)&s->logits_temp, p->vocab_size * sizeof(float));
    s->logits = (float*)malloc(p->vocab_size * sizeof(float));

    // ensure all mallocs went fine
    if (!s->x || !s->xb || !s->xb2 || !s->hb || !s->hb2 || !s->q
        || !s->k || !s->v || !s->logits || !s->key_cache
        || !s->value_cache || !s->logits_gpu) {
        printf("malloc failed!\n");
        exit(1);
    }
}

void free_run_state(RunState* s) {
    hipFree(s->x);
    hipFree(s->xb);
    hipFree(s->xb2);
    hipFree(s->hb);
    hipFree(s->hb2);
    hipFree(s->q);
    hipFree(s->k);
    hipFree(s->v);
    hipFree(s->logits_gpu);
    hipFree(s->logits_temp);
    free(s->logits);
    hipFree(s->key_cache);
    hipFree(s->value_cache);
}

void malloc_weights(TransformerWeights* w, Config* p, int shared_weights) {
    hipMalloc((void**)&w->token_embedding_table, p->vocab_size * p->dim * sizeof(half));
    hipMalloc((void**)&w->rms_att_weight, p->n_layers * p->dim * sizeof(half));
    hipMalloc((void**)&w->rms_ffn_weight, p->n_layers * p->dim * sizeof(half));
    hipMalloc((void**)&w->wq, p->n_layers * p->dim * p->dim * sizeof(half));
    hipMalloc((void**)&w->wk, p->n_layers * p->dim * p->dim * sizeof(half));
    hipMalloc((void**)&w->wv, p->n_layers * p->dim * p->dim * sizeof(half));
    hipMalloc((void**)&w->wo, p->n_layers * p->dim * p->dim * sizeof(half));
    hipMalloc((void**)&w->w1, p->n_layers * p->hidden_dim * p->dim * sizeof(half));
    hipMalloc((void**)&w->w2, p->n_layers * p->dim * p->hidden_dim * sizeof(half));
    hipMalloc((void**)&w->w3, p->n_layers * p->hidden_dim * p->dim * sizeof(half));
    hipMalloc((void**)&w->rms_final_weight, p->dim * sizeof(half));
    int head_size = p->dim / p->n_heads;
    hipMalloc((void**)&w->freq_cis_real, p->seq_len * head_size / 2 * sizeof(half));
    hipMalloc((void**)&w->freq_cis_imag, p->seq_len * head_size / 2 * sizeof(half));

    if (shared_weights)
        w->wcls = w->token_embedding_table;
    else
        hipMalloc((void**)&w->wcls, p->vocab_size * p->dim * sizeof(half));

    // ensure all mallocs went fine
    if (!w->token_embedding_table || !w->rms_att_weight || !w->rms_ffn_weight
        || !w->wq || !w->wk || !w->wv || !w->wo || !w->w1 || !w->w2 || !w->w3 ||
        !w->rms_final_weight || !w->freq_cis_real || !w->freq_cis_imag || !w->wcls) {
        printf("malloc failed!\n");
        exit(1);
    }
}

void free_weights(TransformerWeights* w, int shared_weights) {
    hipFree(w->token_embedding_table);
    hipFree(w->rms_att_weight);
    hipFree(w->rms_ffn_weight);
    hipFree(w->wq);
    hipFree(w->wk);
    hipFree(w->wv);
    hipFree(w->wo);
    hipFree(w->w1);
    hipFree(w->w2);
    hipFree(w->w3);
    hipFree(w->rms_final_weight);
    hipFree(w->freq_cis_real);
    hipFree(w->freq_cis_imag);
    if (!shared_weights)
        hipFree(w->wcls);
}

int divUp(int a, int b) {
    return (a - 1) / b + 1;
}

int uploadWeight(void *w, int elements, FILE* f, void *scratchCpu, void *scratchGpu) {
    int count = fread(scratchCpu, sizeof(float), elements, f);
    if (count != elements) return 1;
    // copy and convert fp32->fp16
    hipMemcpyAsync(scratchGpu, scratchCpu, sizeof(float) * elements, hipMemcpyHostToDevice);
    convert_fp32_to_fp16 <<<divUp(elements, 256), 256 >>> ((half*)w, (float*)scratchGpu, elements);
    return 0;
}

// ----------------------------------------------------------------------------
// initialization: read from checkpoint

int checkpoint_init_weights(TransformerWeights* w, Config* p, FILE* f, int shared_weights) {
    size_t scratch_size = p->n_layers * std::max(p->dim, p->hidden_dim) * p->dim;
    scratch_size = std::max((size_t)p->vocab_size * p->dim, scratch_size);
    scratch_size *= sizeof(float);
    void* scratchCpu = malloc(scratch_size);
    void* scratchGpu = nullptr;
    hipMalloc(&scratchGpu, scratch_size);
    if (uploadWeight(w->token_embedding_table, p->vocab_size * p->dim, f, scratchCpu, scratchGpu)) return 1;
    if (uploadWeight(w->rms_att_weight, p->n_layers * p->dim, f, scratchCpu, scratchGpu)) return 1;
    if (uploadWeight(w->wq, p->n_layers * p->dim * p->dim, f, scratchCpu, scratchGpu)) return 1;
    if (uploadWeight(w->wk, p->n_layers * p->dim * p->dim, f, scratchCpu, scratchGpu)) return 1;
    if (uploadWeight(w->wv, p->n_layers * p->dim * p->dim, f, scratchCpu, scratchGpu)) return 1;
    if (uploadWeight(w->wo, p->n_layers * p->dim * p->dim, f, scratchCpu, scratchGpu)) return 1;
    if (uploadWeight(w->rms_ffn_weight, p->n_layers * p->dim, f, scratchCpu, scratchGpu)) return 1;
    if (uploadWeight(w->w1, p->n_layers * p->dim * p->hidden_dim, f, scratchCpu, scratchGpu)) return 1;
    if (uploadWeight(w->w2, p->n_layers * p->hidden_dim * p->dim, f, scratchCpu, scratchGpu)) return 1;
    if (uploadWeight(w->w3, p->n_layers * p->dim * p->hidden_dim, f, scratchCpu, scratchGpu)) return 1;
    if (uploadWeight(w->rms_final_weight, p->dim, f, scratchCpu, scratchGpu)) return 1;

    int head_size = p->dim / p->n_heads;
    if (uploadWeight(w->freq_cis_real, p->seq_len * head_size / 2, f, scratchCpu, scratchGpu)) return 1;
    if (uploadWeight(w->freq_cis_imag, p->seq_len * head_size / 2, f, scratchCpu, scratchGpu)) return 1;

    if (!shared_weights)
        if (uploadWeight(w->wcls, p->vocab_size * p->dim, f, scratchCpu, scratchGpu)) return 1;

    hipFree(scratchGpu);
    free(scratchCpu);
    return 0;
}

// ----------------------------------------------------------------------------
// neural net blocks

void accum(half* a, half* b, int size) {
    int blocks = divUp(size, 256);
    element_wise_add_kernel << <blocks, 256 >> > (a, b, size);
}


void rmsnorm(half* o, half* x, half* weight, int size) {
    int elementsPerThread = divUp(size, 1024);
    rmsnorm_kernel <<<1, 1024 >>> (o, x, weight, size, elementsPerThread);
}

void softmax(float* x, int size) {
    // find max value (for numerical stability)
    float max_val = x[0];
    for (int i = 1; i < size; i++) {
        if (x[i] > max_val) {
            max_val = x[i];
        }
    }
    // exp and sum
    float sum = 0.0f;
    for (int i = 0; i < size; i++) {
        x[i] = expf(x[i] - max_val);
        sum += x[i];
    }
    // normalize
    for (int i = 0; i < size; i++) {
        x[i] /= sum;
    }
}

void matmul(half* xout, half* x, half* w, int n, int d) {
    int serialElements = divUp(n, 32);
    dim3 block_dim(32, 4);
    int blocks = divUp(d, 4);
    mat_vec_kernel <<<blocks, block_dim >>> (xout, x, w, n, d, serialElements);
}

void RoPERotation(half *q, half *k, half *f_real, half *f_imag, int num_heads, int head_size) {
    RoPERotation_kernel <<<num_heads, head_size / 2 >>> (q, k, f_real, f_imag, num_heads, head_size);
}

void MultiHeadAttention(half *output, half *q, half *key_cache, half *value_cache, int num_heads, int head_size, int loff, int seq_len) {
    int dim = head_size * num_heads;
    MultiHeadAttention_kernel <<<num_heads, 1024>>> (output, q, key_cache, value_cache, num_heads, head_size, loff, seq_len, dim);
}

void siluElementwiseMul(half *hb, half *hb2, int size) {
   silu_element_wise_mul_kernel <<<divUp(size, 256), 256 >>> (hb, hb2, size);
}

void transformer(int token, int pos, Config* p, RunState* s, TransformerWeights* w) {

    // a few convenience variables
    half* x = s->x;
    int dim = p->dim;
    int hidden_dim = p->hidden_dim;
    int head_size = dim / p->n_heads;

    // copy the token embedding into x
    half* content_row = &(w->token_embedding_table[token * dim]);
    hipMemcpyAsync(x, content_row, dim * sizeof(half), hipMemcpyDeviceToDevice);

    // pluck out the "pos" row of freq_cis_real and freq_cis_imag
    half* freq_cis_real_row = w->freq_cis_real + pos * head_size / 2;
    half* freq_cis_imag_row = w->freq_cis_imag + pos * head_size / 2;

    // forward all the layers
    for (int l = 0; l < p->n_layers; l++) {

        // attention rmsnorm
        rmsnorm(s->xb, x, w->rms_att_weight + l * dim, dim);

        // qkv matmuls for this position
        matmul(s->q, s->xb, w->wq + l * dim * dim, dim, dim);
        matmul(s->k, s->xb, w->wk + l * dim * dim, dim, dim);
        matmul(s->v, s->xb, w->wv + l * dim * dim, dim, dim);

        // apply RoPE rotation to the q and k vectors for each head
        RoPERotation(s->q, s->k, freq_cis_real_row, freq_cis_imag_row, p->n_heads, head_size);

        // save key,value at this time step (pos) to our kv cache
        int loff = l * p->seq_len * dim; // kv cache layer offset for convenience
        half* key_cache_row = s->key_cache + loff + pos * dim;
        half* value_cache_row = s->value_cache + loff + pos * dim;
        hipMemcpyAsync(key_cache_row, s->k, dim * sizeof(half), hipMemcpyDeviceToDevice);
        hipMemcpyAsync(value_cache_row, s->v, dim * sizeof(half), hipMemcpyDeviceToDevice);

        MultiHeadAttention(s->xb, s->q, s->key_cache, s->value_cache, p->n_heads, head_size, loff, pos+1);

        // final matmul to get the output of the attention
        matmul(s->xb2, s->xb, w->wo + l * dim * dim, dim, dim);

        // residual connection back into x
        accum(x, s->xb2, dim);

        // ffn rmsnorm
        rmsnorm(s->xb, x, w->rms_ffn_weight + l * dim, dim);

        // Now for FFN in PyTorch we have: self.w2(F.silu(self.w1(x)) * self.w3(x))
        // first calculate self.w1(x) and self.w3(x)
        matmul(s->hb, s->xb, w->w1 + l * dim * hidden_dim, dim, hidden_dim);
        matmul(s->hb2, s->xb, w->w3 + l * dim * hidden_dim, dim, hidden_dim);

        // apply F.silu activation on hb and multiply it with hb2
        siluElementwiseMul(s->hb, s->hb2, hidden_dim);

        // final matmul to get the output of the ffn
        matmul(s->xb, s->hb, w->w2 + l * dim * hidden_dim, hidden_dim, dim);

        // residual connection
        accum(x, s->xb, dim);
    }

    // final rmsnorm
    rmsnorm(x, x, w->rms_final_weight, dim);

    // classifier into logits
    matmul(s->logits_gpu, x, w->wcls, p->dim, p->vocab_size);

    // copy logits from GPU->CPU
    convert_fp16_to_fp32 <<<divUp(p->vocab_size, 256), 256 >>> (s->logits_temp, s->logits_gpu, p->vocab_size);
    hipMemcpy(s->logits, s->logits_temp, p->vocab_size * sizeof(float), hipMemcpyDeviceToHost);
}

// ----------------------------------------------------------------------------
// byte pair encoding (BPE) tokenizer, encodes strings into tokens so we can prompt

int str_lookup(char *str, char **vocab, int vocab_size) {
    // find the first perfect match for str in vocab, return its index or -1 if not found
    for (int i = 0; i < vocab_size; i++) {
        if (strcmp(str, vocab[i]) == 0) {
            return i;
        }
    }
    return -1;
}

void bpe_encode(char *text, char **vocab, float *vocab_scores, int vocab_size, unsigned int max_token_length, int *tokens, int *n_tokens) {
    
    // a temporary buffer to merge two consecutive tokens
    char* str_buffer = (char*) malloc((max_token_length*2+1) * sizeof(char)); // *2 for concat, +1 for null terminator

    // first encode every individual byte in the input string
    *n_tokens = 0; // the number of tokens
    for (char *c = text; *c != '\0'; c++) {
        sprintf(str_buffer, "%c", *c);
        int id = str_lookup(str_buffer, vocab, vocab_size);
        if (id == -1) { printf("not good\n"); exit(1);}
        tokens[*n_tokens] = id;
        (*n_tokens)++;
    }

    // merge the best consecutive pair each iteration, according the scores in vocab_scores
    while (1) {
        float best_score = -1e10;
        int best_id = -1;
        int best_idx = -1;

        for (int i=0; i < (*n_tokens-1); i++) {
            // check if we can merge the pair (tokens[i], tokens[i+1])
            sprintf(str_buffer, "%s%s", vocab[tokens[i]], vocab[tokens[i+1]]);
            int id = str_lookup(str_buffer, vocab, vocab_size);
            if (id != -1 && vocab_scores[id] > best_score) {
                // this merge pair exists in vocab! record its score and position
                best_score = vocab_scores[id];
                best_id = id;
                best_idx = i;
            }
        }

        if (best_idx == -1) {
            break; // we couldn't find any more pairs to merge, so we're done
        }

        // merge the consecutive pair (best_idx, best_idx+1) into new token best_id
        tokens[best_idx] = best_id;
        // delete token at position best_idx+1, shift the entire sequence back 1
        for (int i = best_idx+1; i < (*n_tokens-1); i++) {
            tokens[i] = tokens[i+1];
        }
        (*n_tokens)--; // token length decreased
    }

    free(str_buffer);
}

// ----------------------------------------------------------------------------
// utilities

long time_in_ms() {
    // return time in milliseconds, for benchmarking the model speed
    struct timespec time;
    timespec_get(&time, TIME_UTC);
    return time.tv_sec * 1000 + time.tv_nsec / 1000000;
}

unsigned long long rng_seed;
unsigned int random_u32() {
    // xorshift rng: https://en.wikipedia.org/wiki/Xorshift#xorshift.2A
    rng_seed ^= rng_seed >> 12;
    rng_seed ^= rng_seed << 25;
    rng_seed ^= rng_seed >> 27;
    return (rng_seed * 0x2545F4914F6CDD1Dull) >> 32;
}
float random_f32() { // random float32 in [0,1)
    return (random_u32() >> 8) / 16777216.0f;
}

int sample(float* probabilities, int n) {
    // sample index from probabilities, they must sum to 1
    float r = random_f32();
    float cdf = 0.0f;
    for (int i = 0; i < n; i++) {
        cdf += probabilities[i];
        if (r < cdf) {
            return i;
        }
    }
    return n - 1; // in case of rounding errors
}

int argmax(float* v, int n) {
    // return argmax of v in elements 0..n
    int max_i = 0;
    float max_p = v[0];
    for (int i = 1; i < n; i++) {
        if (v[i] > max_p) {
            max_i = i;
            max_p = v[i];
        }
    }
    return max_i;
}
// ----------------------------------------------------------------------------

int main(int argc, char *argv[]) {

    // poor man's C argparse
    char *checkpoint = NULL;  // e.g. out/model.bin
    float temperature = 0.9f; // e.g. 1.0, or 0.0
    int steps = 256;          // max number of steps to run for, 0: use seq_len
    char *prompt = NULL;      // prompt string

    // 'checkpoint' is necessary arg
    if (argc < 2) {
        printf("Usage: %s <checkpoint_file> [temperature] [steps] [prompt]\n", argv[0]);
        return 1;
    }
    if (argc >= 2) {
        checkpoint = argv[1];
    }
    if (argc >= 3) {
        // optional temperature. 0.0 = (deterministic) argmax sampling. 1.0 = baseline
        temperature = atof(argv[2]);
    }
    if (argc >= 4) {
        steps = atoi(argv[3]);
    }
    if (argc >= 5) {
        prompt = argv[4];
    }

    // seed rng with time. if you want deterministic behavior use temperature 0.0
    rng_seed = (unsigned int)time(NULL);

    // read in the model.bin file
    Config config;
    TransformerWeights weights;
    int shared_weights;
    {
        FILE *file = fopen(checkpoint, "rb");
        if (!file) { printf("Couldn't open file %s\n", checkpoint); return 1; }
        // read in the config header
        if (fread(&config, sizeof(Config), 1, file) != 1) { return 1; }

        // Dump model config
        printf("\nModel params:- \ndim: %d \nhidden_dim: %d\nn_heads: %d\nn_kv_heads: %d\nn_layers: %d\nseq_len: %d\nvocab_size: %d\n\n",
            config.dim, config.hidden_dim, config.n_heads, config.n_kv_heads, config.n_layers, config.seq_len, config.vocab_size);

        // negative vocab size is hacky way of signaling unshared weights. bit yikes.
        shared_weights = config.vocab_size > 0 ? 1 : 0;
        config.vocab_size = abs(config.vocab_size);
        // read in the Transformer weights
        malloc_weights(&weights, &config, shared_weights);
        if (checkpoint_init_weights(&weights, &config, file, shared_weights)) { return 1; }
    }
    // right now we cannot run for more than config.seq_len steps
    if (steps <= 0 || steps > config.seq_len) { steps = config.seq_len; }

    // read in the tokenizer.bin file
    char** vocab = (char**)malloc(config.vocab_size * sizeof(char*));
    float* vocab_scores = (float*)malloc(config.vocab_size * sizeof(float));
    unsigned int max_token_length;
    {
        FILE *file = fopen("tokenizer.bin", "rb");
        if (!file) { printf("couldn't load tokenizer.bin\n"); return 1; }
        if (fread(&max_token_length, sizeof(int), 1, file) != 1) { printf("failed read\n"); return 1; }
        int len;
        for (int i = 0; i < config.vocab_size; i++) {
            if (fread(vocab_scores + i, sizeof(float), 1, file) != 1) { printf("failed read\n"); return 1;}
            if (fread(&len, sizeof(int), 1, file) != 1) { printf("failed read\n"); return 1; }
            vocab[i] = (char *)malloc(len + 1);
            if (fread(vocab[i], len, 1, file) != 1) { printf("failed read\n"); return 1; }
            vocab[i][len] = '\0'; // add the string terminating token
        }
        fclose(file);
    }

    // create and init the application RunState
    RunState state;
    malloc_run_state(&state, &config);

    // process the prompt, if any
    int *prompt_tokens = NULL;
    int num_prompt_tokens = 0;
    if (prompt != NULL) {
        prompt_tokens = (int*)malloc(config.seq_len * sizeof(int));
        bpe_encode(prompt, vocab, vocab_scores, config.vocab_size, max_token_length, prompt_tokens, &num_prompt_tokens);
    }

    // start the main loop
    long start = 0;  // used to time our code, only initialized after first iteration
    int next;        // will store the next token in the sequence
    int token = 1;   // init with token 1 (=BOS), as done in Llama-2 sentencepiece tokenizer
    int pos = 0;     // position in the sequence
    printf("<s>\n"); // explicit print the initial BOS token for stylistic symmetry reasons
    while (pos < steps) {

        // forward the transformer to get logits for the next token
        transformer(token, pos, &config, &state, &weights);

        if(pos < num_prompt_tokens) {
            // if we are still processing the input prompt, force the next prompt token
            next = prompt_tokens[pos];
        } else {
            // sample the next token
            if (temperature == 0.0f) {
                // greedy argmax sampling: take the token with the highest probability
                next = argmax(state.logits, config.vocab_size);
            } else {
                // apply the temperature to the logits
                for (int q=0; q<config.vocab_size; q++) { state.logits[q] /= temperature; }
                // apply softmax to the logits to get the probabilities for next token
                softmax(state.logits, config.vocab_size);
                // we sample from this distribution to get the next token
                next = sample(state.logits, config.vocab_size);
            }
        }

        // following BOS token (1), sentencepiece decoder strips any leading whitespace (see PR #89)
        char *token_str = (token == 1 && vocab[next][0] == ' ') ? vocab[next]+1 : vocab[next];
        printf("%s", token_str);
        fflush(stdout);

        if (next == 2) break; // break if EOS token is reached

        // advance forward
        token = next;
        pos++;
        // init our timer here because the first iteration could be slow
        if (start == 0) { start = time_in_ms(); }
    }

    // report achieved tok/s
    long end = time_in_ms();
    double time = (end - start) / 1000.0;
    int timed_tokens = pos - 1;
    printf("\nachieved tok/s: %f. Tokens: %d, seconds: %g\n", timed_tokens / time, timed_tokens, time);

    // memory cleanup
    free_run_state(&state);
    free_weights(&weights, shared_weights);
    for (int i = 0; i < config.vocab_size; i++) { free(vocab[i]); }
    free(vocab);
    free(vocab_scores);
    if (prompt_tokens != NULL) free(prompt_tokens);
    return 0;
}
